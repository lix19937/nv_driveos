#include "hip/hip_runtime.h"
/**************************************************************
 * @Copyright: 2021-2022 Copyright
 * @Author: lix
 * @Date: 2022-01-10 13:35:47
 * @Last Modified by: lix
 * @Last Modified time: 2022-01-14 14:32:53
 **************************************************************/

#include <hip/hip_fp16.h>
#include <stdint.h>
#include "opencv2/opencv.hpp"

// https://www.fourcc.org/pixel-format/yuv-i420/
// I420 -> It comprises an NxN Y plane followed by (N/2)x(N/2) U and V planes
// is ThreePlaneYUV2BGR

// y skip 2 line,  uv skip 1 line
// for (int j=rangeBegin; j <rangeEnd; j+=2, y1 += stride*2, u1 += width/2,  v1
// += width/2){
//   auto row1 = dst_data + dst_step * j;
//   auto row2 = row1     + dst_step;

//   const auto y2 = y1 + stride;

//   // x access half w
//   for (int i = 0; i <width/2; i+=1, row1 += 3*2, row2 += 3*2){
//     auto u = u1[i];
//     auto v = v1[i];

//     auto vy01 = y1[2 * i];
//     auto vy11 = y1[2 * i + 1];
//     auto vy02 = y2[2 * i];
//     auto vy12 = y2[2 * i + 1];

//     yuv42xxp2rgb8<bIdx, dcn, true>(u, v, vy01, vy11, vy02, vy12, row1, row2);
//   }
// }

// Coefficients for RGB to YUV420p conversion

// R = 1.164(Y - 16) + 1.596(V - 128)
// G = 1.164(Y - 16) - 0.813(V - 128) - 0.391(U - 128)
// B = 1.164(Y - 16)                  + 2.018(U - 128)
//                   ===>
// R = (1220542(Y - 16) + 1673527(V - 128)                  + (1 << 19)) >> 20
// G = (1220542(Y - 16) - 852492(V - 128) - 409993(U - 128) + (1 << 19)) >> 20
// B = (1220542(Y - 16)                  + 2116026(U - 128) + (1 << 19)) >> 20

namespace {

__device__ __forceinline__ void uvToRGBuv(const uint8_t u, const uint8_t v, int& ruv, int& guv, int& buv) {
  const int ITUR_BT_601_CUB = 2116026;
  const int ITUR_BT_601_CUG = -409993;
  const int ITUR_BT_601_CVG = -852492;
  const int ITUR_BT_601_CVR = 1673527;
  const int ITUR_BT_601_SHIFT = 20;
  const int offset = (1 << (ITUR_BT_601_SHIFT - 1));

  int uu = int(u) - 128;
  int vv = int(v) - 128;
  ruv = offset + ITUR_BT_601_CVR * vv;
  guv = offset + ITUR_BT_601_CVG * vv + ITUR_BT_601_CUG * uu;
  buv = offset + ITUR_BT_601_CUB * uu;
}

template <int dcn>
__device__ __forceinline__ void yRGBuvToRGBA(
    const uint8_t vy, const int ruv, const int guv, const int buv, uint8_t& r, uint8_t& g, uint8_t& b, uint8_t& a) {
  const int ITUR_BT_601_CY = 1220542;
  const int ITUR_BT_601_SHIFT = 20;

  int y = max(0, vy - 16) * ITUR_BT_601_CY;

  auto saturate_cast = [](const int& v) { return (uint8_t)((unsigned)v <= 255 ? v : v > 0 ? 255 : 0); };

  r = saturate_cast((y + ruv) >> ITUR_BT_601_SHIFT);
  g = saturate_cast((y + guv) >> ITUR_BT_601_SHIFT);
  b = saturate_cast((y + buv) >> ITUR_BT_601_SHIFT);

  // follow is comment
  if (dcn == 4) {
    a = uint8_t(0xff);
  }
}

template <int bIdx, int dcn, bool is420>
__device__ __forceinline__ void yuv42xxp2rgb8(
    const uint8_t u,
    const uint8_t v,
    const uint8_t vy01,
    const uint8_t vy11,
    const uint8_t vy02,
    const uint8_t vy12,
    uint8_t* row1,
    uint8_t* row2) {
  int ruv, guv, buv;
  uvToRGBuv(u, v, ruv, guv, buv);

  uint8_t r00, g00, b00, a00, r01, g01, b01, a01;
  yRGBuvToRGBA<dcn>(vy01, ruv, guv, buv, r00, g00, b00, a00);
  yRGBuvToRGBA<dcn>(vy11, ruv, guv, buv, r01, g01, b01, a01);

  row1[2 - bIdx] = r00;
  row1[1] = g00;
  row1[bIdx] = b00;
  if (dcn == 4) {
    row1[3] = a00;
  }

  row1[dcn + 2 - bIdx] = r01;
  row1[dcn + 1] = g01;
  row1[dcn + 0 + bIdx] = b01;
  if (dcn == 4) {
    row1[7] = a01;
  }

  if (is420) {
    uint8_t r10, g10, b10, a10, r11, g11, b11, a11;
    yRGBuvToRGBA<dcn>(vy02, ruv, guv, buv, r10, g10, b10, a10);
    yRGBuvToRGBA<dcn>(vy12, ruv, guv, buv, r11, g11, b11, a11);

    row2[2 - bIdx] = r10;
    row2[1] = g10;
    row2[bIdx] = b10;
    if (dcn == 4) {
      row2[3] = a10;
    }

    row2[dcn + 2 - bIdx] = r11;
    row2[dcn + 1] = g11;
    row2[dcn + 0 + bIdx] = b11;
    if (dcn == 4) {
      row2[7] = a11;
    }
  }
}
} // namespace

template <int bIdx, int uIdx, int yIdx, int dcn>
__global__ void yuv422p2rgb_kernel_v1(
    uint8_t* __restrict__ dst_data, const uint8_t* __restrict__ src_data, const int h, const int w) {
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= w / 2 || j >= h) {
    return;
  }

  // const int uidx = 1 - yIdx + uIdx * 2;
  // const int vidx = (2 + uidx) % 4;

  // int dst_step = dcn * w;
  // int src_step = 2 * w; // scn =2

  auto step_iter = src_step * j;

  auto row = dst_data + (dcn * w) * j + (dcn * 2) * i;

  // 0 2 1 3
  // auto i_iter = i * 4;
  // auto u = src_data[step_iter + i_iter + uidx];
  // auto v = src_data[step_iter + i_iter + vidx];
  // auto vy0 = src_data[step_iter + i_iter + yIdx];
  // auto vy1 = src_data[step_iter + i_iter + yIdx + 2];
  // yuv42xxp2rgb8<bIdx, dcn, false>(u, v, vy0, vy1, 0, 0, row, (uint8_t*)(0));

  // uchar4
  uchar4 uyvy = __ldg((uchar4*)src_data + step_iter / 4 + i);
  yuv42xxp2rgb8<bIdx, dcn, false>(uyvy.x, uyvy.z, uyvy.y, uyvy.w, 0, 0, row, (uint8_t*)(0));
}

int yuv422p2rgb(
    uint8_t* dst_data, /* rgb /bgr */
    int h,
    int w,
    const uint8_t* src_data) {
  int block_w = 1, block_h = 1;
  dim3 grid((w / 2 + block_w - 1) / block_w, (h + block_h - 1) / block_h);
  dim3 block(block_w, block_h);

  yuv422p2rgb_kernel_v1<0, 0, 1, 3><<<grid, block, 0, 0>>>(dst_data, src_data, h, w);

  hipPeekAtLastError();
  hipStreamSynchronize(0);
  return 0;
}

int main() {
  auto img_bgr = cv::imread("debug.png");

  int w = img_bgr.cols;
  int h = img_bgr.rows;
  int c = img_bgr.channels();

  ////// Convert from BGR to YUV, Just to get YUV422 data
  cv::Mat img_yuv422(h, w, CV_8UC2, cv::Scalar::all(0));
  cv::cvtColor(img_bgr, img_yuv422, cv::COLOR_BGR2YUV_UYVY);

  ////// YUV422 TO BGR
  cv::Mat img_bgr_gt;
  cv::cvtColor(img_yuv422, img_bgr_gt, cv::COLOR_YUV2BGR_UYVY);
  cv::imwrite("img_bgr_gt.png", img_bgr_gt);

  ////// Byself
  cv::Mat img_bgr_ref(h, w, CV_8UC3, cv::Scalar::all(0));

  //
  void *d_src, *d_dst;
  hipMalloc(&d_src, h * 2 * w);
  hipMalloc(&d_dst, h * 3 * w);

  int nbytes = h * 2 * w;
  hipMemcpy(d_src, img_yuv422.data, nbytes, ::hipMemcpyHostToDevice);

  yuv422p2rgb((uint8_t*)d_dst, h, w, (const uint8_t*)d_src);

  nbytes = h * 3 * w;
  hipMemcpy(img_bgr_ref.data, d_dst, nbytes, ::hipMemcpyDeviceToHost);

  cv::imwrite("img_bgr_ref.png", img_bgr_ref);

  hipFree(d_src);
  hipFree(d_dst);

  return 0;
}

/*
export LD_LIBRARY_PATH=$LD_LIBRARY_PATH:/mnt/d/workspace/opencv-4.9.0/build/install_local/lib
nvcc -std=c++14 -O2 ./yuv422p2rgb.cu  -lopencv_core -lopencv_imgproc -lopencv_highgui -lopencv_imgcodecs \
-I/mnt/d/workspace/opencv-4.9.0/build/install_local/include/opencv4 \
-L/mnt/d/workspace/opencv-4.9.0/build/install_local/lib


*/
